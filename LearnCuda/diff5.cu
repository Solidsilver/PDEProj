#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <cstdlib>
#include <iostream>

#define WIDTH 1000
#define HEIGHT 1000
#define DEPTH 2
#define A 0
#define B 1
#define dT 1

using namespace std;
using namespace sf;


__constant__ float weights[3][3] = {
    {0.05, 0.2, 0.05}, {0.2, -1, 0.2}, {0.05, 0.2, 0.05}};

__device__ float diffA(int x, int y) { return 1; }

__device__ float diffB(int x, int y) { return 0.5; }

__device__ float feed(int x, int y) { return 0.0367; }

__device__ float kill(int x, int y) { return 0.0649; }

// Translate indicies of 3d array index to flattened 1d array
__device__ int trIdx(int i, int j, int k) {
  return i * WIDTH * DEPTH + j * DEPTH + k;
}

__device__ int trIdx2(int i, int j, int k, int width, int depth) {
  return i * width * depth + j * depth + k;
}

__device__ float conv(float *arr, int x, int y, int z) {
  float sum = 0;
  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      sum += weights[i + 1][j + 1] * arr[trIdx(y + i, x + j, z)];
    }
  }
  return sum;
}

__device__ float reactA(float valA, float valB, int x, int y) {
  return feed(x, y) * (1 - valA) - valA * valB * valB;
}

__device__ float reactB(float valA, float valB, int x, int y) {
  return valA * valB * valB - (kill(x, y) + feed(x, y)) * valB;
}

__device__ float initA(int x, int y) { return 1; }

__device__ float initB(int x, int y) {
  if (y > HEIGHT / 2 - 200 && y < HEIGHT / 2 + 200 && x > WIDTH / 2 - 200 &&
      x < WIDTH / 2 + 200) {
    return 1;
  }
  return 0;
}

__global__ void init_state(float *arr) {
  // Setup GPU thread indexing
  int tidxI = blockIdx.x;
  int strideI = gridDim.x;
  int tidxJ = threadIdx.x;
  int strideJ = blockDim.x;

  for (int i = tidxI; i < HEIGHT; i += strideI) {
    for (int j = tidxJ; j < WIDTH; j += strideJ) {
      arr[trIdx(i, j, A)] = initA(j, i);
      arr[trIdx(i, j, B)] = initB(j, i);
    }
  }
}

__global__ void update(float *cur, float *prev) {
  // Setup GPU thread indexing
  int tidxI = blockIdx.x;
  int strideI = gridDim.x;
  int tidxJ = threadIdx.x;
  int strideJ = blockDim.x;

  for (int i = tidxI + 1; i < HEIGHT - 1; i += strideI) {
    for (int j = tidxJ + 1; j < WIDTH - 1; j += strideJ) {
      float prevA = prev[trIdx(i, j, A)];
      float prevB = prev[trIdx(i, j, B)];

      float curA =
          prevA +
          (diffA(j, i) * conv(prev, j, i, A) + reactA(prevA, prevB, j, i)) * dT;
      float curB =
          prevB +
          (diffB(j, i) * conv(prev, j, i, B) + reactB(prevA, prevB, j, i)) * dT;

      cur[trIdx(i, j, A)] = curA;
      cur[trIdx(i, j, B)] = curB;
    }
  }
}


__global__ void fill_pixels(Uint8 *pixels, float *arr) {
  int indexY = blockIdx.x;
  int strideY = gridDim.x;
  int indexX = threadIdx.x;
  int strideX = blockDim.x;
  for (int i = indexY; i < HEIGHT; i += strideY) {
    for (int j = indexX; j < WIDTH; j += strideX) {
      int indexA = trIdx(i, j, A);
      int indexB = trIdx(i, j, B);
      // int comb = arr[indexA] + arr[indexB];
      // if (comb < 0) {
      //   comb *= -1;
      // }
      pixels[trIdx2(i, j, 0, WIDTH, 4)] = 0 * 255;
      pixels[trIdx2(i, j, 1, WIDTH, 4)] = (arr[indexB]) * 255;
      pixels[trIdx2(i, j, 2, WIDTH, 4)] = (arr[indexA]) * 255;
      pixels[trIdx2(i, j, 3, WIDTH, 4)] = 255;
    }
  }
}

int main() {

  // cuda settings
  int blockSize = 1024;
  int numBlocks = HEIGHT;
  bool paused = true;

  // Create the window
  sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Reaction-Diffusion");
  // window.setFramerateLimit(10);

  sf::Event event;

  sf::Texture texture;
  texture.create(WIDTH, HEIGHT);
  sf::IntRect r(0, 0, WIDTH, HEIGHT);
  sf::Sprite sprite(texture, r);

  // Uint8 *pixels = new Uint8[WIDTH * HEIGHT * 4];
  Uint8 *pixels;
  Uint8 *pixLocal = new Uint8[WIDTH * HEIGHT * 4];
  hipMalloc((void **)&pixels, WIDTH * HEIGHT * 4);

  // Setup cuda
  float *cur;
  float *prev;
  hipMalloc((void **)&cur, HEIGHT * WIDTH * 2 * sizeof(float));
  hipMalloc((void **)&prev, HEIGHT * WIDTH * 2 * sizeof(float));
  init_state<<<numBlocks, blockSize>>>(cur);
  init_state<<<numBlocks, blockSize>>>(prev);

  // Show initial state
  fill_pixels<<<numBlocks, blockSize>>>(pixels, prev);
  hipDeviceSynchronize();
  hipMemcpy(pixLocal, pixels, WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
  texture.update(pixLocal);
  window.draw(sprite);

  // Start the anim. loop
  while (window.isOpen()) {
    while (window.pollEvent(event)) {
      if (event.type == sf::Event::Closed) {
        window.close();
      }
      if (event.type == Event::KeyPressed &&
          event.key.code == Keyboard::Space) {
        paused = !paused;
      }
    }

    if (!paused) {
      update<<<numBlocks, blockSize>>>(cur, prev);
      fill_pixels<<<numBlocks, blockSize>>>(pixels, cur);
      hipMemcpy(pixLocal, pixels, WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost);
      hipDeviceSynchronize();

      texture.update(pixLocal);
      window.draw(sprite);

      float *tmp = cur;
      cur = prev;
      prev = tmp;
    }

    // Update the window
    window.display();
  }

  // Free mem
  hipFree(cur);
  hipFree(prev);
  hipFree(pixels);
  free(pixLocal);

  return 0;
}
