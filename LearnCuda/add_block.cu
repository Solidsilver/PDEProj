
#include <hip/hip_runtime.h>
#include <__clang_cuda_builtin_vars.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void) {
  int N = 1 << 20; // 1M elements

  float *x, *y;
  // Allocate unified memory in the GPU (accesible in CPU or GPU)
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on elements on GPU
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free mem in GPU
  hipFree(x);
  hipFree(y);

  return 0;
}